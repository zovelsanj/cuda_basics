
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel (int* a) {
    a[-1] = 1;
}

void check_error(hipError_t err) {
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

int main() {
    int* a;
    hipError_t err = hipMalloc(&a, -sizeof(int));
    check_error(err);

    kernel<<<1, -1>>>(a);
    err = hipGetLastError();
    check_error(err);

    err = hipDeviceSynchronize();
    check_error(err);

    free(a);
}
