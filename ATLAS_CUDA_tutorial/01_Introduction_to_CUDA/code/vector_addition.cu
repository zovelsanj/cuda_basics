
#include <hip/hip_runtime.h>
#define N 512

__global__ void add(int* a, int* b, int* c) {

    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];

}

int main() {

    int *a, *b, *c;           // host copies of a, b, c
    int *d_a, *d_b, *d_c;     // device copies of a, b, c

    int size = N * sizeof(int);

    // Alloc space for device copies of a, b, c
    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    // Alloc space for host copies of a, b, c and setup input values
    a = (int *) malloc(size); memset(a, 0, size);
    b = (int *) malloc(size); memset(b, 0, size);
    c = (int *) malloc(size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU with N blocks
    add<<<N,1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;

}
