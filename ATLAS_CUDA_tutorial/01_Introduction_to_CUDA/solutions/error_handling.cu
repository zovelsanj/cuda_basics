
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel (int* a) {
    a[0] = 1;
}

int main() {
    hipError_t err;

    int* a;
    err = hipMalloc(&a, sizeof(int));
    if (err != hipSuccess) {
        printf("CUDA error %s\n", hipGetErrorString(err));
        exit(-1);
    }

    kernel<<<1, 1>>>(a);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error %s\n", hipGetErrorString(err));
        exit(-1);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error %s\n", hipGetErrorString(err));
        exit(-1);
    }

    err = hipFree(a);
    if (err != hipSuccess) {
        printf("CUDA error %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
