
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void set_value (int* a) {
    *a = 2;
}

int main() {
    int* a;
    int* d_a;

    // Allocate the host copy of a
    a = (int*) malloc(sizeof(int));
    // Allocate the device copy of a
    // By convention, device copies of 
    // variables are often prefixed with d_
    hipMalloc(&d_a, sizeof(int));

    // Set the host value of a
    *a = 1;

    // Copy the value of a to the device
    hipMemcpy(d_a, a, sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel to set the value
    set_value<<<1,1>>>(d_a);
    hipDeviceSynchronize();

    // Copy the value of a back to the host
    hipMemcpy(a, d_a, sizeof(int), hipMemcpyDeviceToHost);
    
    // Check that the value of a is correct
    if (*a == 2) {
        printf("Success!\n");
    }
    else {
        printf("Failure\n");
    }
    
    // Clean up a
    free(a);
    // Clean up d_a
    hipFree(d_a);
}
