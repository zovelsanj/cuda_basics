
#include <hip/hip_runtime.h>
#include <stdio.h>

const int DSIZE = 32*1048576;

// error checking macro
#define cudaCheckErrors(msg)                                    \
    do {                                                        \
        hipError_t __err = hipGetLastError();                 \
        if (__err != hipSuccess) {                             \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",  \
                    msg, hipGetErrorString(__err),             \
                    __FILE__, __LINE__);                        \
            fprintf(stderr, "*** FAILED - ABORTING\n");         \
            exit(1);                                            \
        }                                                       \
    } while (0)

__global__ void kernel(float *x, int ds) {
    for (int idx = threadIdx.x+blockDim.x*blockIdx.x; idx < ds; idx+=gridDim.x*blockDim.x) {
        float result = 1.0 + x[idx];

        #pragma unroll 1000               // Force the compiler to unroll the loop into 100 sequential statements
        for (int j = 1; j <= 1000; ++j) {
            result *= (float) j;
        }
        x[idx] = result;
    }
}

int main() {
    float *x;

    hipMalloc(&x, DSIZE*sizeof(float));
    cudaCheckErrors("hipMalloc failure");
    hipMemset(x, 0, DSIZE*sizeof(float)); // initialize array to zero;
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    
    // Use maximum threads per block

    int blocks = 4096;  // modify this line for experimentation
    int threads = 1024; // modify this line for experimentation
    
    // int threads = prop.maxThreadsPerBlock;
    // int blocks = (DSIZE + threads - 1) / threads;
    
    kernel<<<blocks, threads>>>(x, DSIZE);
    cudaCheckErrors("kernel launch failure");

    hipDeviceSynchronize();
    cudaCheckErrors("kernel execution failure");

    return 0;
}
